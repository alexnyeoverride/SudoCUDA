#include "hip/hip_runtime.h"
﻿#include <array>
#include <iostream>
#include <hip/hip_runtime.h>
#include "array.cuh"
#include <cstdint>

#define _ (-1)
using Board = std::array<std::array<int8_t, 9>, 9>;

// Each unit of work is a board with all values except one known, plus a guess.
struct Work {
	Board board;
	int8_t guessRow;
	int8_t guessColumn;
};


__device__ __host__ void makeGuesses(
	Board* workingBoard,
	DeviceArray<Work> workStealingStack
) {
	for (int8_t row = 0; row < 9; ++row) {
		for (int8_t col = 0; col < 9; ++col) {
			if ((*workingBoard)[row][col] == _) {
				for (int8_t guess = 1; guess <= 9; ++guess) {
					Board newBoard = *workingBoard;
					newBoard[row][col] = guess;

					// TODO: threads could deadlock here.  Maybe split threads into solvers and guessers / poppers and pushers.
					//		Make makeGuesses __global__ and from solve dispatch it as its own thread.

					Result<Work> result = {Error::Overflow};
					do {
						result = workStealingStack.push({
							newBoard,
							row,
							col
						});
					} while (result.error == Error::Overflow);
				}
			}
		}
	}
}

__device__ bool applyConstraints(
	const Work* work
) {
	const int8_t column = work->guessColumn;
	const int8_t row = work->guessRow;
	const auto workingBoard = work->board;

	// Apply constraint to row
	for (int8_t i = 0; i < 9; i++) {
		if (i == column) {
			continue;
		}
		if (workingBoard[row][i] == workingBoard[row][column]) {
			return false;
		}
	}
	// Apply constraint to column
	for (int8_t i = 0; i < 9; i++) {
		if (i == row) {
			continue;
		}
		if (workingBoard[i][column] == workingBoard[row][column]) {
			return false;
		}
	}
	// Apply constraint to box
	const int8_t boxStartI = row / 3 * 3;
	const int8_t boxStartJ = column / 3 * 3;
	for (int8_t i = boxStartI; i < boxStartI + 3; i++) {
		for (int8_t j = boxStartJ; j < boxStartJ + 3; j++) {
			if (row == i && column == j) {
				continue;
			}
			if (workingBoard[i][j] == workingBoard[row][column]) {
				return false;
			}
		}
	}

	return true;
}

__device__ bool complete(const Board* board) {
	for (int8_t i = 0; i < 9; i++) {
		for (int8_t j = 0; j < 9; j++) {
			if ((*board)[i][j] == _) {
				return false;
			}
		}
	}
	return true;
}

__global__ void solve(
	Board* knownValues,
	DeviceArray<Work> workStealingStack,
	const int* numWorkingThreads
) {
	while (true) {
		if (complete(knownValues)) {
			break;
		}

		auto [error, work] = workStealingStack.pop();
		if (error == Error::Underflow) {
			if (*numWorkingThreads == 0) {
				break; // If workStealingStack is empty and no threads are working, the puzzle is unsolvable.
			}
			continue; // Wait for work.
		}

		if (!applyConstraints(&work)) {
			continue; // This guess is invalid.  Look for new work.
		}

		if (complete(&work.board)) {
			*knownValues = work.board;
		}

		// TODO: work path permutation optimization would go here.
		//		If the work guess path is a permutation of a prior guess path, skip it.
		//		Advanced optimization, need a hash set of sorted position-value pair set keys.

		makeGuesses(knownValues, workStealingStack);
	}
}


int main() {
	Board* knownValues;
	hipMallocManaged(&knownValues, sizeof(Board));

	// TODO: what capacity makes sense?  How do I measure the impact of threads' waiting to insert work?
	auto workStealingStack = HostArray<Work>(1024*1024*100);

	int* numWorkingThreads;
	hipMallocManaged(&numWorkingThreads, sizeof(int));
	*numWorkingThreads = 0;

	constexpr Board puzzle = {
		8, _, _, _, _, _, _, _, _,
		_, _, 3, 6, _, _, _, _, _,
		_, 7, _, _, 9, _, 2, _, _,
		_, 5, _, _, _, 7, _, _, _,
		_, _, _, _, 4, 5, 7, _, _,
		_, _, _, 1, _, _, _, 3, _,
		_, _, 1, _, _, _, _, 6, 8,
		_, _, 8, 5, _, _, _, 1, _,
		_, 9, _, _, _, _, 4, _, _
	};
	*knownValues = puzzle;

	// Initialize workStealingStack to the first layer of guesses for all empty cells.
	makeGuesses(knownValues, workStealingStack);

	solve<<<1, 32>>>(knownValues, workStealingStack, numWorkingThreads);
	if (const hipError_t err = hipGetLastError(); err != hipSuccess) {
		std::cerr << hipGetErrorString(err) << std::endl;
	}
}