#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <stdgpu/vector.cuh>

static void printErrors() {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cerr << hipGetErrorString(err) << std::endl;
	}
}

__global__ void kernel(int* v, int* n, int cap) {
	atomicAdd(n, 1);
	if (*n < cap) {
		v[*n] = 42;
	}
}

int main() {
	// dynamic array.
	int* v;
	int* n;
	int cap = 2;

	hipMallocManaged(reinterpret_cast<void **>(&n), sizeof(int));
	hipMallocManaged(reinterpret_cast<void **>(&v), sizeof(int) * cap);

	kernel<<<1, 1>>>(v, n, cap);
	hipDeviceSynchronize();
	printErrors();

	std::cout << v[1] << std::endl;

	hipFree(v);
	return 0;
}